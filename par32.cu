#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

typedef unsigned long ulint;
typedef unsigned long long ulint64;

int banyakdata = 256000;
int dimensigrid = 2000;
int dimensiblok = 128;

__host__ __device__ void modexp(ulint a, ulint b, ulint c, ulint* res) {
	ulint64 s = a;
	ulint64 ans = 1;
	while (b != 0) {
		if (b % 2 == 1) {
			ans = ans * s % c;
			b--;
		}
		b /= 2;
		if (b != 0) {
			s = s * s %c;
		}
	}
	*res = ans;
}

__device__ void enkripsi(ulint g, ulint k, ulint p, ulint m, ulint y, ulint *res) {
	modexp(g, k, p, res);
	modexp(y, k, p, res + 1);
	
	*(res + 1) = *(res + 1) * m % p;
}

__device__ void dekripsi(ulint a, ulint b, ulint p, ulint e, ulint *res) {
	modexp(a, e, p, res);
	*res = *res * b % p;
}

__global__ void kernelenk(ulint *m, ulint *k, ulint g, ulint p, ulint y, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	enkripsi(g, k[i], p, m[i], y, res + 2 * i);
}

__global__ void kerneldek(ulint *c, ulint p, ulint e, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	dekripsi(c[2*i], c[2*i+1], p, e, res + i);
}

hipError_t enkripsiCUDA(ulint *m, ulint *k, ulint g, ulint p, ulint y, ulint *res) {
	hipError_t cudaStatus;

	hipSetDevice(0);

	//=====================BAGIAN M[] K[] DAN RES[] ====================================//
	ulint *devm, *devk, *devres;
	


	hipMalloc((void**)&devm, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devk, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devres, banyakdata * 2 * sizeof(ulint));
	
	hipMemcpy((devm), m, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);
	hipMemcpy((devk), k, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);	

		// size_t free_byte ;
  //       size_t total_byte ;
  //       hipMemGetInfo( &free_byte, &total_byte ) ;
		// double free_db = (double)free_byte ;
  //       double total_db = (double)total_byte ;
  //       double used_db = total_db - free_db ;

  //       printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

  //           used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

	// printf("<<<<<<<<<<<<<<<<<<KERNEL>>>>>>>>>>>>>>>>>\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernelenk << <dimensigrid, dimensiblok>> >(devm,devk,g,p,y,devres);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Durasi enkripsi= %f ms\n", milliseconds);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else {
		//printf("Success\n");
	}

	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res, devres, (sizeof(ulint) * 2 * banyakdata), hipMemcpyDeviceToHost);


	Error:

	
	hipFree(devm);
	hipFree(devk);
	hipFree(devres);

	return cudaStatus;
}

hipError_t dekripsiCUDA(ulint *c, ulint p, ulint e, ulint *res2) {
	hipError_t cudaStatus;

	hipSetDevice(0);

	//=====================BAGIAN M[] K[] DAN RES[] ====================================//
	ulint *devc, *devres2;
	

	hipMalloc((void**)&devc, banyakdata * 2 * sizeof(ulint));
	hipMalloc((void**)&devres2, banyakdata * sizeof(ulint));
	
	hipMemcpy((devc), c, (sizeof(ulint) * banyakdata * 2), hipMemcpyHostToDevice);

	// printf("<<<<<<<<<<<<<<<<<<KERNEL>>>>>>>>>>>>>>>>>\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kerneldek << <dimensigrid, dimensiblok>> >(devc,p,e,devres2);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Durasi dekripsi= %f ms\n", milliseconds);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else {
		//printf("Success\n");
	}

	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res2, devres2, (sizeof(ulint) * banyakdata), hipMemcpyDeviceToHost);


	Error:

	
	hipFree(devc);
	hipFree(devres2);

	return cudaStatus;

}

void initenkripsi(ulint *m, ulint *k){
	
	for (int i = 0; i < banyakdata; i++) {
		m[i] = rand() % 3999999978;
		k[i] = rand() % 3999999978;
	}	
}

int main(){
	ulint *m, *k, *res, *res2, g, p, y, x, e;

	m = (ulint*)malloc(banyakdata * sizeof(ulint));
	k = (ulint*)malloc(banyakdata * sizeof(ulint));
	res = (ulint*)malloc(banyakdata * 2 * sizeof(ulint));
	res2 = (ulint*)malloc(banyakdata * sizeof(ulint));

	srand(2018);

	g = rand() % 3999999978;
	p = 3999999979;
	x = rand() % 3999999978;
	modexp(g,x,p,&y);
	initenkripsi(m, k);

	// printf("<<<<<<<<<<<<<<Pesan Asli>>>>>>>>>>>>>>>\n");
	// for (int i = 0; i < 4; i++) {
	// 	printf("m[%d] = %lu\n", i, m[i]);
	// }

	// printf("m[...]\n");
	// printf("m[%d] = %lu\n", banyakdata-1, m[banyakdata-1]);

	enkripsiCUDA(m,k,g,p,y,res);

	// printf("<<<<<<<<<<<<<<Hasil Enkripsi>>>>>>>>>>>>>>>\n");
	// for (int i = 0; i < 4; i++) {
	// 	printf("c[%d] = %lu 	c[%d] = %lu\n", 2*i, res[2*i], 2*i+1, res[2*i+1]);
	// }

	// printf("c ...\n");
	// printf("c[%d] = %lu 	c[%d] = %lu\n", banyakdata * 2-2, res[banyakdata * 2-2], banyakdata *2-1,res[banyakdata*2-1]);

	e = p-x-1;
	dekripsiCUDA(res,p,e,res2);

	// printf("<<<<<<<<<<<<<<Hasil Dekripsi>>>>>>>>>>>>>>>\n");
	// for (int i = 0; i < 4; i++) {
	// 	printf("m[%d] = %lu\n", i, res2[i]);
	// }

	// printf("m[...]\n");
	// printf("m[%d] = %lu\n", banyakdata-1, res2[banyakdata-1]);

	free(m);
	free(k);
	free(res);
	free(res2);

	return 0;
}
