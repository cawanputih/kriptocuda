#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

typedef unsigned long ulint;
typedef unsigned long long ulint64;

int banyakdata = 10240;
int dimensigrid = 80;
int dimensiblok = 128;

__host__ __device__ void modexp(ulint a, ulint b, ulint c, ulint* res) {
	ulint64 s = a;
	ulint64 ans = 1;
	while (b != 0) {
		if (b % 2 == 1) {
			ans = ans * s % c;
			b--;
		}
		b /= 2;
		if (b != 0) {
			s = s * s %c;
		}
	}
	*res = ans;
}

__device__ void enkripsi(ulint g, ulint k, ulint p, ulint m, ulint y, ulint *res) {
	modexp(g, k, p, res);
	modexp(y, k, p, res + 1);
	
	*(res + 1) = *(res + 1) * m % p;
}

__device__ void dekripsi(ulint a, ulint b, ulint p, ulint e, ulint *res) {
	modexp(a, e, p, res);
	*res = *res * b % p;
}

__global__ void kernelenk(ulint *m, ulint *k, ulint g, ulint p, ulint y, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	enkripsi(g, k[i], p, m[i], y, res + 2 * i);
}

__global__ void kerneldek(ulint *c, ulint p, ulint e, ulint *res) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	dekripsi(c[2*i], c[2*i+1], p, e, res + i);
}

hipError_t enkripsiCUDA(ulint *m, ulint *k, ulint g, ulint p, ulint y, ulint *res) {
	hipError_t cudaStatus;

	hipSetDevice(0);

	//=====================BAGIAN M[] K[] DAN RES[] ====================================//
	ulint *devm, *devk, *devres;
	


	hipMalloc((void**)&devm, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devk, banyakdata * sizeof(ulint));
	hipMalloc((void**)&devres, banyakdata * 2 * sizeof(ulint));
	
	hipMemcpy((devm), m, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);
	hipMemcpy((devk), k, (sizeof(ulint) * banyakdata), hipMemcpyHostToDevice);	

		// size_t free_byte ;
  //       size_t total_byte ;
  //       hipMemGetInfo( &free_byte, &total_byte ) ;
		// double free_db = (double)free_byte ;
  //       double total_db = (double)total_byte ;
  //       double used_db = total_db - free_db ;

  //       printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

  //           used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

	// printf("<<<<<<<<<<<<<<<<<<KERNEL>>>>>>>>>>>>>>>>>\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernelenk << <dimensigrid, dimensiblok>> >(devm,devk,g,p,y,devres);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\nDurasi enkripsi= %f ms\n", milliseconds);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else {
		//printf("Success\n");
	}

	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res, devres, (sizeof(ulint) * 2 * banyakdata), hipMemcpyDeviceToHost);


	Error:

	
	hipFree(devm);
	hipFree(devk);
	hipFree(devres);

	return cudaStatus;
}

hipError_t dekripsiCUDA(ulint *c, ulint p, ulint e, ulint *res2) {
	hipError_t cudaStatus;

	hipSetDevice(0);

	//=====================BAGIAN M[] K[] DAN RES[] ====================================//
	ulint *devc, *devres2;
	

	hipMalloc((void**)&devc, banyakdata * 2 * sizeof(ulint));
	hipMalloc((void**)&devres2, banyakdata * sizeof(ulint));
	
	hipMemcpy((devc), c, (sizeof(ulint) * banyakdata * 2), hipMemcpyHostToDevice);

	// printf("<<<<<<<<<<<<<<<<<<KERNEL>>>>>>>>>>>>>>>>>\n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kerneldek << <dimensigrid, dimensiblok>> >(devc,p,e,devres2);

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\nDurasi enkripsi= %f ms\n", milliseconds);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else {
		//printf("Success\n");
	}

	hipDeviceSynchronize();

	//	COPY FROM DEVICE TO HOST HERE 
	hipMemcpy(res2, devres2, (sizeof(ulint) * banyakdata), hipMemcpyDeviceToHost);


	Error:

	
	hipFree(devc);
	hipFree(devres2);

	return cudaStatus;

}

void initenkripsi(ulint *m, ulint *k) {
	for (int i = 0; i < banyakdata; i++)
    {
        m[i] = 0;
    }

	FILE *file = fopen("plain.plain", "r");
    char *code;
    size_t n = 0;
    int c;

    code = (char*) malloc(9999999);
    
    while ((c = fgetc(file)) != EOF)
    {
        code[n++] = (char) c;
    }
    code[n] = '\0';

 
    char karakter = code[0];

    int i = 0;
    int indexpesan = -1;
    while(karakter != '\0'){
        karakter = code[i];
        if(i % 3== 0){
            indexpesan++;
            m[indexpesan] += karakter * 1000000;
        }else if(i % 3 ==1){
            m[indexpesan] += karakter * 1000;
        }else{
            m[indexpesan] += karakter;
        }
        i++;
    }

    // printf("count : %d\n", indexpesan);

    // nilai k //
	srand(2018);

	for (int i = 0; i < banyakdata; i++) {
		k[i] = rand() % 3999999978;
	}
}

ulint stringtolong(char* s){
	ulint res = 0;
	int i = 0;
	while(s[i] != '\0'){
		res *= 10;
		res += s[i] - '0';
		i++;
	}
	return res;
}

void initdekripsi(ulint *c) {
	for (int i = 0; i < banyakdata*2; i++)
	{
		c[i] = 0;
	}

	char *buffer = 0;
	long length;
	FILE *f = fopen("cipher.cipher", "rb");

	if (f)
	{
		fseek(f, 0, SEEK_END);
		length = ftell(f);
		fseek(f, 0, SEEK_SET);
		buffer = (char*)malloc(length);
		if (buffer) {
			fread(buffer, 1, length, f);
		}
		buffer[length] = '\0';
		fclose(f);
	}
	char delimstrip[2];
	delimstrip[0] = 45;
	delimstrip[1] = 0;

	// Baca seluruh ciphertext
	char *tempsplit;
	tempsplit = strdup(strtok(buffer, delimstrip));
	c[0] = stringtolong(tempsplit);
	tempsplit = strdup(strtok(NULL, delimstrip));
	c[1] = stringtolong(tempsplit);
	// Baca m
	for (int i = 1; i < banyakdata; i++) {
		tempsplit = strdup(strtok(NULL, delimstrip));
		c[2*i] = stringtolong(tempsplit);
		tempsplit = strdup(strtok(NULL, delimstrip));
		c[2*i+1] = stringtolong(tempsplit);
	}
}

void initenkripsi2(ulint *m, ulint *k){
	
	for (int i = 0; i < banyakdata; i++) {
		m[i] = rand() % 3999999978;
		k[i] = rand() % 3999999978;
	}	
}

void writecipher(ulint* c){
	FILE *fp = fopen("cipher.cipher","w");
	

	for (int i = 0; i < banyakdata*2; i++)
	{
		fprintf(fp, "%lu", c[i]);
		fprintf(fp, "%c", '-');
	}

	fclose(fp);
}

void writedekrip(ulint* m){
	FILE *fp = fopen("dekrip.dekrip","w");
	

	for (int i = 0; i < banyakdata; i++)
	{
		ulint temp = m[i];
		fprintf(fp, "%c",  temp/1000000 );
		fprintf(fp, "%c",  (temp/1000) % 1000 );
		fprintf(fp, "%c",  temp % 1000);
	}

	fclose(fp);
}

int main(){
	ulint *m, *k, *res, *res2, g, p, y, x, e, *res3;

	m = (ulint*)malloc(banyakdata * sizeof(ulint));
	k = (ulint*)malloc(banyakdata * sizeof(ulint));
	res = (ulint*)malloc(banyakdata * 2 * sizeof(ulint));
	res2 = (ulint*)malloc(banyakdata * sizeof(ulint));
	res3 = (ulint*)malloc(banyakdata * 2 *sizeof(ulint));

	srand(2018);

	g = rand() % 3999999978;
	p = 3999999979;
	x = rand() % 3999999978;
	modexp(g,x,p,&y);
	initenkripsi(m, k);
	// initenkripsi2(m, k);


	printf("<<<<<<<<<<<<<<Pesan Asli>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("m[%d] = %lu\n", i, m[i]);
	}

	printf("m[...]\n");
	printf("m[%d] = %lu\n", banyakdata-1, m[banyakdata-1]);

	enkripsiCUDA(m,k,g,p,y,res);

	printf("<<<<<<<<<<<<<<Hasil Enkripsi>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("c[%d] = %lu 	c[%d] = %lu\n", 2*i, res[2*i], 2*i+1, res[2*i+1]);
	}

	printf("c ...\n");
	printf("c[%d] = %lu 	c[%d] = %lu\n", banyakdata * 2-2, res[banyakdata * 2-2], banyakdata *2-1,res[banyakdata*2-1]);

	writecipher(res);

	initdekripsi(res3);

	e = p-x-1;
	dekripsiCUDA(res3,p,e,res2);

	printf("<<<<<<<<<<<<<<Hasil Dekripsi>>>>>>>>>>>>>>>\n");
	for (int i = 0; i < 4; i++) {
		printf("m[%d] = %lu\n", i, res2[i]);
	}

	printf("m[...]\n");
	printf("m[%d] = %lu\n", banyakdata-1, res2[banyakdata-1]);
	writedekrip(res2);

	free(m);
	free(k);
	free(res);
	free(res2);

	return 0;
}